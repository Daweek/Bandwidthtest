#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <unistd.h>
#include <sys/time.h>
#include <cutil.h>
#include <cutil_inline.h>

#define MAXDEV 32
#define NLOOP 2e11
#define PKG	  1024
#define MAXSIZE 1024*1024*600
static const double MEGA  = 1e6;
static const double MICRO = 1e-6;

static void
get_cputime(double *nowp, double *deltap)
{
    struct timeval t;
    double now0;

    gettimeofday(&t, NULL);
    now0 = t.tv_sec + t.tv_usec/1000000.0;
    *deltap = now0 - *nowp;
    *nowp   = now0;
}

static void
sendperf(int argc, char **argv)
{
    int maxsize = MAXSIZE;
    int i, j;
    size_t size;
    double sized;
    double now = 0.0, dt = 0.0;
    double ratio = 2;
    double nloop = NLOOP;
    char *src[MAXDEV];
    char *dst[MAXDEV];
    int ndev;

    ndev = 1; // !!!

    printf("# %d device%s found.\n", ndev, ndev > 1 ? "s" : "");
    for (i = 0; i < ndev; i++) {
        hipSetDevice(i);
        cutilSafeCall(hipMalloc((void**) &dst[i], sizeof(char) * maxsize));
	src[i] = (char *)malloc(sizeof(char) * maxsize);
    }
    printf("\n#\n# hipMemcpy (HostToDevice)\n#\n");

    for (sized = 1024; sized < maxsize; sized *= ratio) {

        size = (size_t)sized;

	get_cputime(&now, &dt);
	for (j = 0; j < nloop/size; j++) {
	    for (i = 0; i < ndev; i++) {
  	        hipMemcpy(dst[i], src[i], size, hipMemcpyHostToDevice);
	    }
	}
        hipDeviceSynchronize();
	get_cputime(&now, &dt);


	  printf("%d byte    %f sec    %f MB/s\n", size, dt, nloop/MEGA/dt);

    }


}

static void
receiveperf(int argc, char **argv)
{
    int maxsize = MAXSIZE;
    int i, j;
    size_t size;
    double sized;
    double now = 0.0, dt = 0.0;
    double ratio = 2;
    double nloop = NLOOP;
    char *src[MAXDEV];
    char *dst[MAXDEV];
    int ndev;


    ndev = 1; // !!!

    printf("# %d device%s found.\n", ndev, ndev > 1 ? "s" : "");
    for (i = 0; i < ndev; i++) {

        cutilSafeCall(hipMalloc((void**) &src[i], sizeof(char) * maxsize));
	dst[i] = (char *)malloc(sizeof(char) * maxsize);
    }
    printf("\n#\n# hipMemcpy (DeviceToHost)\n#\n");


    for (sized = 1024; sized < maxsize; sized *= ratio) {

    	size = (size_t)sized;

		get_cputime(&now, &dt);
		for (j = 0; j < nloop/size; j++) {
			for (i = 0; i < ndev; i++) {
				hipSetDevice(i);
					hipMemcpy(dst[i], src[i], size, hipMemcpyDeviceToHost);
		}
	}

	hipDeviceSynchronize();
	get_cputime(&now, &dt);
	printf("%d byte    %f sec    %f MB/s\n",size, dt, nloop/MEGA/dt);
	}
}

int main(int argc, char **argv)
{
	printf("Starting Bandwidth Test...\n");
    sendperf(argc, argv);
    receiveperf(argc, argv);

    fprintf(stderr, "going to quit...\n");
    return 0;
}
